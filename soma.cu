#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ 
void get(int *output)
{
    *output = 10;;
}

int main(int argc, char *argv[])
{
    int *h_num;
    int *d_num;    
    hipError_t cudaerr;

    h_num = (int*)malloc(sizeof(int));

    cudaerr = hipMalloc((void **) &d_num, sizeof(int));
    if (cudaerr != hipSuccess)
        printf("nao pode alocar memoria no device\n");

    get<<<1, 1>>>(d_num);

    cudaerr = hipMemcpy(h_num, d_num, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaerr != hipSuccess)
        printf("nao pode copiar memoria\n");

    printf("%d\n", *h_num);

    hipFree(d_num);
    return 0;
}

